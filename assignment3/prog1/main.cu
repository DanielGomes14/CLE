#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>

#include "common.h"

#include "cmd/processCommandLine.cuh"
#include "utils/utils.cuh"


/**
 * @brief Host processing logic, column by column.
 *
 * @param order order of the matrices
 * @param amount amount of matrices
 * @param matrixArray array with matrices
 * @param results array to store matrices determinants
 */
void hostCC(int order, int amount, double **matrixArray, double *results);

/**
 * @brief Device processing logic, column by column.
 *
 * @param d_matrixArray pointer to matrices' array.
 * @param amount amount of matrices
 * @param order order of matrices
 * @param results pointer to store matrices determinants
 * @return __global__
 */
__global__ void deviceCC(double *d_matrixArray, double *d_results);
// void deviceCC(int order, int amount, double **matrixArray, double *results);

/**
 * @brief Main logic of the program.
 * Makes gaussian elimination on the host and the device.
 * Compares the obtained results at the end.
 *
 * @param argc amount of arguments in the command line
 * @param argv array with the arguments from the command line
 * @return int return execution status of operation
 */
int main(int argc, char **argv)
{
    // process command line information to obtain file names
    int fileAmount = 0;
    char **fileNames;
    if (processInput(argc, argv, &fileAmount, &fileNames))
    {
        perror("error processing input");
        exit(EXIT_FAILURE);
    }

    // device setup
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // process files
    double *h_matrixArray = NULL;
    int order = 0, amount = 0;
    for (int i = 0; i < fileAmount; i++)
    {
        // read data from file
        readData(*(fileNames + i), &h_matrixArray, &order, &amount);

        // structure to save results
        double *retrieved_results = (double *)malloc(sizeof(double) * amount);

        // allocate memory on device
        double *d_matrixArray;
        double *d_results;
        CHECK(hipMalloc((void **)&d_matrixArray, (sizeof(double) * order * order * amount)));
        CHECK(hipMalloc((void **)&d_results, sizeof(double) * amount));

        // copy data to device memory
        CHECK(hipMemcpy(d_matrixArray, h_matrixArray, (sizeof(double) * order * order * amount), hipMemcpyHostToDevice));

        // create grid and block
        dim3 grid(amount, 1, 1);
        dim3 block(order, 1, 1);

        // DEVICE PROCESSING
        double d_start = seconds();

        deviceCC<<<grid, block>>>(d_matrixArray, d_results);

        CHECK(hipDeviceSynchronize());
        double drr = seconds() - d_start;
        printf("Device processing took <%.5f> seconds.\n", drr);

        CHECK(hipGetLastError());                                                                        // check kernel errors
        CHECK(hipMemcpy(retrieved_results, d_results, sizeof(double) * amount, hipMemcpyDeviceToHost)); // return obtained results
        CHECK(hipFree(d_matrixArray));                                                                   // free device memory

        // HOST PROCESSING
        double h_results[amount];
        double start = seconds();
        hostCC(order, amount, &h_matrixArray, h_results);
        double hrr = seconds() - start;
        printf("Host processing took <%.5f> seconds.\n", hrr);

        printf("\nRESULTS\n");
        for (int i = 0; i < amount; i++)
        {
            printf("HOST: <%+5.3e>\t DEVICE: <%+5.3e>\n", h_results[i], retrieved_results[i]);
        }
    }

    return 0;
}

/**
 * @brief Calculates determinant column by column on the host
 *
 * @param matrix pointer to matrix
 * @param order order of matrix
 * @return int determinant of matrix
 */
void hostCC(int order, int amount, double **matrixArray, double *results)
{
    for (int i = 0; i < amount; i++)
    {
        *(results + i) = column_by_column_determinant(order, ((*matrixArray) + (i * order * order)));
        // printf("%+5.3e\n", *(results + i));
    }
}

/**
 * @brief Calculates the determinant column by column on the GPU
 *
 * @param d_matrixArray the array with all matrices
 * @param d_results the array to store the results
 * @return __global__
 */
__global__ void deviceCC(double *d_matrixArray, double *d_results)
{
    
    int order = blockDim.x; /** The order of the Matrix is the size of the block since a block is a matrix **/ 
    int matrixIdx = blockIdx.x * order * order; /** jump to the current matrix**/
    int tColumn = threadIdx.x + matrixIdx; /** The column for which each thread is responsible**/
    int pivotColumn; /**  Auxiliar variable to select the index column of the pivot **/
   
    for (int currElem = 0; currElem < order; currElem++)
    {

        if (threadIdx.x < currElem)
            return;

        int iterColumn = currElem + matrixIdx; /** jump to the column of the matrix of the current iteration **/

        double pivot = d_matrixArray[iterColumn + currElem * order];  /** The Pivot will be initially the diagonal element on the iterColumn**/
        pivotColumn = iterColumn; 

        // Only one thread should do the Partial Pivoting, and update the determinant value of the Matrix
        if (threadIdx.x == currElem)
        {
            // iterate through the remaining columns of the same row
            for (int col = iterColumn + 1; col < ( matrixIdx + order); ++col)
            {   

                // if there's a bigger value on the row than the current pivot, the choosen pivot will be updated
                if (fabs(d_matrixArray[(currElem * order) + col]) > fabs(pivot))
                {

                    // update the value of the pivot and pivot col index
                    pivot = d_matrixArray[(currElem * order) + col];
                    pivotColumn = col;
                   
                }
            }
            
            if (currElem == 0)
                // initialize the results index of the matrix on the first iteration 
                d_results[blockIdx.x] = 1;

            // if the elected pivot is different from the initial one, than we perform a swap on the columns
            if (pivotColumn != iterColumn)
            {
               
                for (int k = 0; k < order; k++)
                {
                    double temp;
                    temp = d_matrixArray[(k * order) + iterColumn];
                    d_matrixArray[(k * order) + iterColumn] = d_matrixArray[(k * order) + pivotColumn];
                    d_matrixArray[(k * order) + pivotColumn] = temp;
                }
                d_results[blockIdx.x] *= -1.0; // signal the row swapping
            }

            d_results[blockIdx.x] *= pivot;
            return;
            //continue;
        }
        // syncronize all threads in the current block https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
        __syncthreads();
        iterColumn = currElem + matrixIdx;
        pivot = d_matrixArray[iterColumn + currElem * order];
        pivotColumn = iterColumn;

        // perform the reduction of the base matrix
        double const_val = d_matrixArray[tColumn + order * currElem] / pivot;
        for (int row = currElem + 1; row < order; row++)
        {

            d_matrixArray[tColumn + order * row] -= d_matrixArray[pivotColumn + order * row] * const_val;
        }
        __syncthreads();
    }
}

