#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <time.h>

#include "common.h"

#include "cmd/processCommandLine.cuh"
#include "utils/utils.cuh"

/**
 *   program configuration
 */
#ifndef SECTOR_SIZE
# define SECTOR_SIZE  512
#endif
#ifndef N_SECTORS
# define N_SECTORS    (1 << 21)                            // it can go as high as (1 << 21)
#endif

/**
 * @brief Host processing logic, row by row.
 * 
 * @param order order of the matrices
 * @param amount amount of matrices
 * @param matrixArray array with matrices
 * @param results array to store matrices determinants
 */
void hostRR(int order, int amount, double **matrixArray, double *results);

/**
 * @brief Device processing logic, row by row.
 * 
 * @param d_matrixArray pointer to matrices' array.
 * @param amount amount of matrices
 * @param order order of matrices
 * @param results pointer to store matrices determinants
 * @return __global__ 
 */
__global__ void deviceRR(double *d_matrixArray, double *d_results);
// void deviceRR(int order, int amount, double **matrixArray, double *results);

/**
 * @brief Main logic of the program. 
 * Makes gaussian elimination on the host and the device.
 * Compares thre obtained results at the end. 
 * 
 * @param argc amount of arguments in the command line
 * @param argv array with the arguments from the command line
 * @return int return execution status of operation
 */
int main(int argc, char **argv)
{
    // process command line information to obtain file names
    int fileAmount = 0;
    char ** fileNames;
    if(processInput(argc, argv, &fileAmount, &fileNames))
    {
        perror("error processing input");
        exit(EXIT_FAILURE);
    }

    // device setup
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // process files
    double *h_matrixArray = NULL;
    int order = 0, amount = 0;
    for(int i = 0; i < fileAmount; i++)
    {
        // read data from file
        readData(*(fileNames + i), &h_matrixArray, &order, &amount);

        // for(int j = 0; j < amount; j++)
        // {
        //     printf("%f\t", *(h_matrixArray + j));
        //     break;
        // }

        // structure to save results
        double *retrieved_results = (double *)malloc(sizeof(double) * amount);

        // allocate memory on device
        double *d_matrixArray;
        double *d_results;
        CHECK(hipMalloc((void **)&d_matrixArray, (sizeof(double) * order * order * amount)));
        CHECK(hipMalloc((void **)&d_results, sizeof(double) * amount));

        // copy data to device memory
        CHECK(hipMemcpy(d_matrixArray, h_matrixArray, (sizeof(double) * order * order * amount), hipMemcpyHostToDevice));

        // create grid and block
        dim3 grid(amount, 1, 1);
        dim3 block(order, 1, 1); 

        // DEVICE PROCESSING
        double d_start = seconds();
        deviceRR<<<grid, block>>>(d_matrixArray, d_results);
        CHECK (hipDeviceSynchronize ());
        double drr = seconds() - d_start;
        printf("Device processing took <%.5f> seconds.\n", drr);

        CHECK(hipGetLastError ());         // check kernel errors
        CHECK(hipMemcpy(retrieved_results, d_results, sizeof(double) * amount, hipMemcpyDeviceToHost));   // return obtained results
        CHECK(hipFree (d_matrixArray));    // free device memory

        // HOST PROCESSING
        double h_results[amount];
        double start = seconds();
        hostRR(order, amount, &h_matrixArray, h_results);
        double hrr = seconds() - start;
        printf("Host processing took <%.5f> seconds.\n", hrr);

        printf("\nRESULTS\n");
        for(int i = 0; i < amount; i++)
        {
            printf("HOST: <%+5.3e>\t DEVICE: <%+5.3e>\n", h_results[i], retrieved_results[i]);
        }
    }

    return 0;
}

/**
 * @brief Calculates determinant row by row
 * 
 * @param matrix pointer to matrix
 * @param order order of matrix
 * @return int determinant of matrix
 */
void hostRR(int order, int amount, double **matrixArray, double *results)
{
    for(int i = 0; i < amount; i++)
    {
        *(results + i) = row_by_row_determinant(order, ((*matrixArray) + (i * order * order)));
        // printf("%+5.3e\n", *(results + i));
    }
}

/**
 * @brief 
 * 
 * @param d_matrixArray 
 * @param d_results 
 * @return __global__ 
 */
__global__ void deviceRR(double *d_matrixArray, double *d_results)
{
    int n = blockDim.x;

    for(int iter = 0; iter < n; iter++)
    {
        if(threadIdx.x < iter)
            continue;

        int matrixIdx = blockIdx.x * n * n;
        int row = matrixIdx + threadIdx.x * n;
        int iterRow = matrixIdx + iter * n;
        
        if(threadIdx.x == iter)
        {
            if(iter == 0)
            {
                d_results[blockIdx.x] = 1;
            }
            d_results[blockIdx.x] *= d_matrixArray[iterRow + iter];
            continue;
        }

        double pivot = d_matrixArray[iterRow + iter];

        double value = d_matrixArray[row + iter] / pivot;
        for(int i = iter + 1; i < n; i++)
        {
            d_matrixArray[row + i] -= d_matrixArray[iterRow + i] * value;
        }
        __syncthreads();
    }
}
